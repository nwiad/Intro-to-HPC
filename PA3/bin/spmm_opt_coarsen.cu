#include "hip/hip_runtime.h"
#include "spmm_opt.h"
#include <cstdio>

// sparse matrix: (num_v, ?), dense matrix: (?, INFEATURE), output: (num_v, INFEATURE)
// use shared memory and 1d thread block

#define BLOCK_SIZE 16
#define WARP_SIZE 32
#define MASK 0xffffffff
#define COARSEN_FACTOR 2

__global__ void spmm_kernel_opt(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int feat_in) {
    __shared__ int sm_k[BLOCK_SIZE][WARP_SIZE];
    __shared__ float sm_v[BLOCK_SIZE][WARP_SIZE];
    int tid_x = threadIdx.x >> 5;
    int tid_y = threadIdx.x & 0x1f;
    int i = blockIdx.x * BLOCK_SIZE + tid_x;
    int j = blockIdx.y * WARP_SIZE * COARSEN_FACTOR + tid_y;
    if (i >= num_v || j >= feat_in) return;
    int begin = ptr[i], end = ptr[i + 1];

    float res[COARSEN_FACTOR];
    #pragma unroll
    for (int t = 0; t < COARSEN_FACTOR; t++) {
        res[t] = 0.0;
    }

    for (int p = begin; p < end; p += WARP_SIZE) {
        if (p + tid_y < end) {
            sm_k[tid_x][tid_y] = idx[p + tid_y];
            sm_v[tid_x][tid_y] = val[p + tid_y];
        }
        else {
            sm_k[tid_x][tid_y] = 0;
            sm_v[tid_x][tid_y] = 0.0;
        }
        __syncwarp();
        #pragma unroll
        for (int kk = 0; kk < WARP_SIZE; kk++) {
            int col = sm_k[tid_x][kk];
            float value = sm_v[tid_x][kk];
            #pragma unroll
            for (int t = 0; t < COARSEN_FACTOR; t++) {
                if (j + t * WARP_SIZE >= feat_in) break;
                res[t] += value * vin[col * feat_in + j + t * WARP_SIZE];
            }
        }
    }
    #pragma unroll
    for (int t = 0; t < COARSEN_FACTOR; t++) {
        if (j + t * WARP_SIZE >= feat_in) break;
        vout[i * feat_in + j + t * WARP_SIZE] = res[t];
    }
}

void SpMMOpt::preprocess(float *vin, float *vout)
{
    // TODO: your code
    grid.x = (num_v + BLOCK_SIZE - 1) / BLOCK_SIZE;
    grid.y = (feat_in + WARP_SIZE * COARSEN_FACTOR - 1) / (WARP_SIZE * COARSEN_FACTOR);
    block.x = BLOCK_SIZE * WARP_SIZE;
}

void SpMMOpt::run(float *vin, float *vout)
{
    // TODO: your code
    spmm_kernel_opt<<<grid, block>>>(d_ptr, d_idx, d_val, vin, vout, num_v, feat_in);
}